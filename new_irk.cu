#include "hip/hip_runtime.h"
//to run  - nvcc new_irk.cu  -lcublas -lcurand -lcusparse  -lcusolver -lcudart  -o  rg7

#include <typeinfo> // for usage of C++ typeid
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h>

#include "hipsolver.h"
#include <hipsolver.h>
#include "hipsolver.h"
#include "hipblas.h"

#include <hip/hip_runtime_api.h>
#include "hipsparse.h"
#include <iostream>
using namespace std;
  
//profiling the code
#define GPUERRCHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define TIME_INDIVIDUAL_LIBRARY_CALLS

#define DBICGSTAB_MAX_ULP_ERR   100
#define DBICGSTAB_EPS           1.E-14f //9e-2

#define CLEANUP()                       \
do {                                    \
    if (x)          free (x);           \
    if (f)          free (f);           \
    if (r)          free (r);           \
    if (rw)         free (rw);          \
    if (p)          free (p);           \
    if (pw)         free (pw);          \
    if (s)          free (s);           \
    if (t)          free (t);           \
    if (v)          free (v);           \
    if (tx)         free (tx);          \
    if (Aval)       free(Aval);         \
    if (AcolsIndex) free(AcolsIndex);   \
    if (ArowsIndex) free(ArowsIndex);   \
    if (Mval)       free(Mval);         \
    if (devPtrX)    checkCudaErrors(hipFree (devPtrX));                    \
    if (devPtrF)    checkCudaErrors(hipFree (devPtrF));                    \
    if (devPtrR)    checkCudaErrors(hipFree (devPtrR));                    \
    if (devPtrRW)   checkCudaErrors(hipFree (devPtrRW));                   \
    if (devPtrP)    checkCudaErrors(hipFree (devPtrP));                    \
    if (devPtrS)    checkCudaErrors(hipFree (devPtrS));                    \
    if (devPtrT)    checkCudaErrors(hipFree (devPtrT));                    \
    if (devPtrV)    checkCudaErrors(hipFree (devPtrV));                    \
    if (devPtrAval) checkCudaErrors(hipFree (devPtrAval));                 \
    if (devPtrAcolsIndex) checkCudaErrors(hipFree (devPtrAcolsIndex));     \
    if (devPtrArowsIndex) checkCudaErrors(hipFree (devPtrArowsIndex));     \
    if (devPtrMval)       checkCudaErrors(hipFree (devPtrMval));           \
    if (stream)           checkCudaErrors(hipStreamDestroy(stream));       \
    if (cublasHandle)     checkCudaErrors(hipblasDestroy(cublasHandle));     \
    if (cusparseHandle)   checkCudaErrors(hipsparseDestroy(cusparseHandle)); \
    fflush (stdout);                                    \
} while (0)

using namespace std;
#define BLOCK_SIZE 32

extern "C" int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }


void gpu_blas_mmul(const double *A, const double*B, double *C, const int m, const int k, const int n) {
  int lda=m,ldb=k,ldc=m;
  const double alf = 1;
  const double bet = 0;
  const double *alpha = &alf;
  const double *beta = &bet;

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Do the actual multiplication
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
  // Destroy the handle
  hipblasDestroy(handle);
}

__global__ void copy_kernel(const double * __restrict d_in1, double * __restrict d_out1, const double * __restrict d_in2, double * __restrict d_out2, const int M, const int N) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < N) && (j < N)) {
        d_out1[j * N + i] = d_in1[j * M + i];
        d_out2[j * N + i] = d_in2[j * M + i];
    }
}

int bicg(double h_A1_dense[] ,double x0[] , double B[] , double M[] , int  rmaxit , double rtol, int n ){
double *r0 = (double*)malloc(n*n*sizeof(double ));
double *r0_tilde = (double*)malloc(n*n*sizeof(*r0_tilde));
double *r = (double*)malloc(n*n*sizeof(double));
double *r_temp_tilde = (double*)malloc(n*n*sizeof(*r_temp_tilde));
double *z = (double*)malloc(n*sizeof(*z));
double *z_tilde = (double*)malloc(n*sizeof(*z_tilde));
double *p = (double*)malloc(n*sizeof(*p));
double *p_tilde = (double*)malloc(n*sizeof(*p_tilde));
double *q = (double*)malloc(n*sizeof(*q));
double *q_tilde = (double*)malloc(n*sizeof(*q_tilde));
double normr=0;double normb=0; int tot_iter=0;
double resid;
double tol,betta;
int work_size=0;
double tempp=0;
// r=b-Ax;
double *h_A_trans= (double*)malloc(n*n*sizeof(double));
for(int i=0;i<n;i++)for(int j=0;j<n;j++){
  h_A_trans[i*n+j]=h_A1_dense[j*n+i];
}



// C=Ax
hipblasHandle_t Blas_handle;
hipblasCreate(&Blas_handle); 
hipsolverHandle_t solver_handle;
hipsolverDnCreate(&solver_handle);

int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;


double *h_C=(double*)malloc(n*sizeof(*h_C));
nr_rows_A=n;nr_cols_A=n;nr_rows_B=n;nr_cols_B=1;nr_rows_C=n;nr_cols_C=1;

double *d_rA1;hipMalloc(&d_rA1,nr_rows_A * nr_cols_A * sizeof(*d_rA1));
double *d_B;hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(*d_B));
double *d_C;hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(*d_C));
//int d_rA1_ColIndices ;hipMalloc(&d_rA1_ColIndices, nnzA * sizeof(*d_rA1_ColIndices));
hipMemcpy(d_rA1,h_A1_dense,nr_rows_A * nr_cols_A * sizeof(double),hipMemcpyHostToDevice);
hipMemcpy(d_B,B,nr_rows_B * nr_cols_B * sizeof(double),hipMemcpyHostToDevice);

gpu_blas_mmul(d_rA1, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(double),hipMemcpyDeviceToHost);
for(int i=0;i<n;i++){r[i]=B[i]-h_C[i];r_temp_tilde[i]=r[i];r0[i]=r[i];r0_tilde[i]=r[i];    normr+=r[i]*r[i];normb+=B[i]*B[i];}
normr=sqrt(normr);normb=sqrt(normb);
if(!normb)normb=1;
resid=normr/normb;
if(resid<rtol){
  tol=resid;
  tot_iter=0;
  return 0;

}  
double rho1=0;double rho2=0;

double *d_M; hipMalloc(&d_M, nr_rows_A*nr_cols_A*sizeof(*d_M));
hipMemcpy(d_M, M, nr_rows_A * nr_cols_A * sizeof(double),hipMemcpyHostToDevice );
// CUDA QR initialisation

double *d_TAU; hipMalloc((void **)&d_TAU,min(nr_cols_A,nr_rows_A)*sizeof(double) );
hipsolverDnDgeqrf_bufferSize(solver_handle,nr_rows_A,nr_cols_A,d_M,nr_rows_A,&work_size);
double *work; hipMalloc(&work ,work_size*sizeof(double));

// CUDA GERF exec.
int *dev_info; hipMalloc(&dev_info, sizeof(int));

hipsolverDnDgeqrf(solver_handle, nr_rows_A, nr_cols_A, d_M,nr_rows_A, d_TAU, work, work_size, dev_info);
//hipsolverHandle_t handle, int m, int n, double *A, int lda, double *TAU, double *Workspace, int Lwork, int *devInfo

int dev_info_h=0;hipMemcpy(&dev_info_h,dev_info, sizeof(int ), hipMemcpyDeviceToHost);
if(dev_info_h!=0 )cout<<"uncussful exec of GERf"<<endl;
double *h_Q =(double *)malloc(nr_rows_A*nr_cols_A*sizeof(double));
memset(h_Q, 0,  nr_rows_A*nr_cols_A*sizeof(double));
for(int i=0;i<nr_rows_A;i++)h_Q[i+i*nr_rows_A]=1;
double *d_Q; hipMalloc(&d_Q, nr_rows_A*nr_cols_A*sizeof(double));
hipMemcpy(d_Q,h_Q,nr_rows_A*nr_cols_A*sizeof(double),hipMemcpyHostToDevice);
//CUDA QR execution
hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT,HIPBLAS_OP_N,nr_rows_A,nr_cols_A,
  min(nr_rows_A,nr_cols_A), d_M, nr_rows_A,d_TAU,d_Q,nr_rows_A,work,work_size,dev_info);


// 






  double *d_r; hipMalloc(&d_r, nr_rows_A*nr_cols_A*sizeof(double));

   double *d_p; hipMalloc(&d_p, n*sizeof(double ));

   double *d_rA1_trans;hipMalloc(&d_rA1_trans, n*n*sizeof(double ));
  double *d_R ; hipMalloc(&d_R,nr_cols_A*nr_cols_A*sizeof(double));
  double  *h_Bl= (double *)malloc(nr_cols_A*nr_cols_A*sizeof(double));
  double  *d_Bl; hipMalloc(&d_Bl,nr_cols_A*nr_cols_A*sizeof(double));
double *d_qq; hipMalloc(&d_qq, n*sizeof(double ));

for(int i=0;i<rmaxit; i++){
  // solve Mz=r;//block solve
 // if(i==5 or i==6)cout<<i<<" "<<resid<<" "<<normr<< endl;
// if(i==1){ for(int j=0;j<n;j++){cout<<r0[j]<<" ";}cout<<endl;for(int j=0;j<n;j++){cout<<r0_tilde[j]<<" ";} }
  for(int j=0;j<n;j++){r[j]=r0[j];r_temp_tilde[j]=r0_tilde[j];}
  hipMemcpy(d_r,r,nr_rows_A*nr_cols_A*sizeof(double),hipMemcpyHostToDevice);
  hipsolverDnDormqr(solver_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_T,nr_rows_A,nr_cols_A,min(nr_cols_A,nr_rows_A),
    d_M,nr_rows_A,d_TAU,d_r,nr_rows_A,work,work_size,dev_info);


  // at this point d_r contains the element Q^Tr 
  // only the first coloumn if d_r makes sense ...
  
  hipMemcpy(r,d_r,nr_rows_A*nr_cols_A*sizeof(double ), hipMemcpyDeviceToHost);
  dim3 Grid(iDivUp(nr_cols_A,BLOCK_SIZE),iDivUp(nr_cols_A,BLOCK_SIZE));
  dim3 Block(BLOCK_SIZE,BLOCK_SIZE);
  copy_kernel<<<Grid, Block>>>(d_M,d_R,d_r,d_Bl,nr_rows_A,nr_cols_A);



  // solving an upper triangular linear system
  const double alpha =1;
   hipblasDtrsm(Blas_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,nr_cols_A,nr_cols_A,
                &alpha,d_R,nr_cols_A,d_Bl,nr_cols_A);
   hipMemcpy(h_Bl,d_Bl,nr_cols_A*nr_cols_A*sizeof(double), hipMemcpyDeviceToHost);
    for(int j =0;j<n;j++ ){z[j]=h_Bl[j];if(i==3)cout<<z[j]<<" ";}




      // solve Mtz=r_temp_tilde 
    for(int j =0;j<n;j++)z_tilde[j]=-1*z[j]/1.67;

    rho1=0;
    for(int j=0;j<n;j++ )rho1+=z[j]*r0_tilde[j];
    if(rho1==0){
      tol =normr/normb;
      tot_iter=i;
      return 1;
    }  

    if(i==0){
      

      for(int j=0;j<n;j++){
        p[j]=z[j];p_tilde[j]=z_tilde[j];
      }
    }
    else{
     betta=rho1/rho2;


   
    //if(i==1){cout<<"A is \n"; for(int j=0;j<n;j++){ for(int k=0;k<n;k++) cout<<h_A1_dense[j*n+k]<<" ";  }cout<<endl;          }
    //if(i==1){ cout<<"p is  \n" ;for(int j=0;j<n;j++)cout<<p[j]<<" "; }

     for(int j=0;j<n;j++) {
      p[j]=betta*p[j]+z[j];
      p_tilde[j]=betta*p_tilde[j]+z_tilde[j];
     //if(i==1)cout<<" p[ "<<j<<" ] ="<<p_tilde[j]<<endl;

     }

    // q=Ap and q_tilde =At*p
   
   hipMemcpy(d_p, p, n* sizeof(double),hipMemcpyHostToDevice );

   
   
   gpu_blas_mmul(d_rA1, d_p, d_qq, nr_rows_A, nr_cols_A, nr_cols_B);

   

   hipMemcpy(q, d_qq, n* sizeof(double),hipMemcpyDeviceToHost ); if(i==0)for(int j=0;i<n;j++){cout<<q[j]<<" ";}
   hipMemcpy(d_rA1_trans,h_A_trans,n*n*sizeof(double),hipMemcpyHostToDevice);
   hipMemcpy(d_p, p_tilde, n* sizeof(double),hipMemcpyHostToDevice );

   gpu_blas_mmul(d_rA1_trans, d_p, d_qq, nr_rows_A, nr_cols_A, nr_cols_B);
   hipMemcpy(q_tilde, d_qq, n* sizeof(double),hipMemcpyDeviceToHost );

   tempp=0;
    for(int j=0;j<n;j++){}
   for(int j=0;j<n;j++)tempp+=p_tilde[j]*q[j];
   double alphaa = -1*rho1/tempp;
   for(int j=0;j<n;j++){
      // x0[j]+=alphaa*p[j];
      // r0[j]+=alphaa*q[j];
      // r0_tilde[j]=alphaa*q_tilde[j];
      
      }
      rho2=rho1;
      normr=0;
      for(int j=0;j<n;j++)normr+=r0[j]*r0[j];
        normr=sqrt(normr);resid=normr/normb;
      if(resid<rtol){
          tol=resid;
          tot_iter=i;
          return 3;
      }


    }
    


  
}
tol =resid;

hipFree(d_rA1);
hipFree(d_B);
hipFree(d_C);
hipFree(d_M);

hipFree(d_Q);
hipFree(d_qq);
hipFree(d_TAU);
hipFree(dev_info);

hipFree(d_r);
hipFree(d_R);
hipFree(work);
hipFree(d_Bl);


hipFree(d_rA1_trans);
free(r0);
free(r0_tilde);
free(r_temp_tilde);
free(r);
free(z);
free(z_tilde);
free(q_tilde);
free(q);
free(p);
free(p_tilde);
free(h_A_trans);
free(h_Q);
free(h_C);
free(h_Bl);
hipblasDestroy(Blas_handle);
hipsolverDnDestroy(solver_handle);



return 1;





}

int main(){
const int n=4;const int r=2;
const int N=n;

int rmaxit,max_iter,irka_iter;
double rtol,itol;

double *x0 = (double*)malloc(n*sizeof(double));    

double *x0_tilde= (double*)malloc(n*sizeof(*x0_tilde));  
double *A=(double*)malloc(n*n*sizeof(double));
double *B=(double*)malloc(n*sizeof(*B));
double *C=(double*)malloc(n*sizeof(*C)); 
double *res=(double*)malloc(n*sizeof(*res));
double *res_tilde=(double*)malloc(n*sizeof(*res_tilde));
double *sig=(double*)malloc(r*sizeof(*sig));
double *sig_old=(double*)malloc(r*sizeof(*sig_old));
double *temp_v=(double*)malloc(n*sizeof(*temp_v));
double *temp_w=(double*)malloc(n*sizeof(*temp_w));
double *eye_n=(double*)malloc(n*n*sizeof(*eye_n));
double *V =(double *)malloc (n*n*sizeof(double));
double *W =(double *)malloc (n*n*sizeof(double));

  double *B_red=(double *)malloc(r*sizeof(double));
  double *C_red=(double *)malloc(r*sizeof(double));
  double *A_red=(double *)malloc(r*r*sizeof(double));

double error=100007;
max_iter=100;
rmaxit  =100;
rtol= 0.0001;
itol= 0.0001;
srand((unsigned)time(0));
for(int i=0;i<n;i++){
  B[i]=rand()%10;B[i]/=10;
  C[i]=rand()%10;C[i]/=10;
  for(int j=0;j<n;j++){
    if(i==j)eye_n[i*n+j]=1;
    else eye_n[i*n+j]=0;
    double tempx=rand()%10;
    if(tempx>7 or tempx< 3){A[i*n+j]=(rand()%10);}
    else A[i*n+j]=0;
  }
}

//for(int i=0;i<n;i++){for(int j=0;j<n;j++){cout<<eye_n[i*n+j]<<" ";}cout<<endl;}
//initialize sparse matrix A
hipsparseHandle_t handle; hipsparseCreate(&handle);

double *d_A_dense;  hipMalloc(&d_A_dense, n * n * sizeof(double));
double *d_EYE_dense ; hipMalloc(&d_EYE_dense ,n*n*sizeof(double));

hipMemcpy(d_A_dense, A, n * n * sizeof(double), hipMemcpyHostToDevice);
//hipMemcpy(d_A_dense, A , n*n*sizeof(double),hipMemcpyHostToDevice);
hipMemcpy(d_EYE_dense, eye_n , n*n*sizeof(double),hipMemcpyHostToDevice);

hipsparseMatDescr_t descrA;    hipsparseCreateMatDescr(&descrA);
hipsparseSetMatType    (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);  



hipsparseMatDescr_t descrEYE; hipsparseCreateMatDescr(&descrEYE);
hipsparseSetMatType    (descrEYE, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(descrEYE, HIPSPARSE_INDEX_BASE_ZERO);
int nnzA = 0;             // --- Number of nonzero elements in dense matrix A

const int lda = N;
int nnzEYE=0;
  int *d_nnzPerVectorA;   hipMalloc(&d_nnzPerVectorA, n * sizeof(*d_nnzPerVectorA));
  hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, n, n, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA);



cout<<"nnzA is equal to "<<nnzA<<endl;
//int *d_nnzPerVectorA; hipMalloc(&d_nnzPerVectorA,n*sizeof(*d_nnzPerVectorA));
//hipsparseDnnz(handle,HIPSPARSE_DIRECTION_ROW,n,n,descrA,d_A_dense,lda,d_nnzPerVectorA, &nnzA);


int *d_nnzPerVectorEYE; hipMalloc(&d_nnzPerVectorEYE,n*sizeof(*d_nnzPerVectorEYE));
hipsparseDnnz(handle,HIPSPARSE_DIRECTION_ROW,n,n,descrEYE,d_EYE_dense,lda,d_nnzPerVectorEYE, &nnzEYE);


int *h_nnzPerVectorA = (int *)malloc(n * sizeof(*h_nnzPerVectorA));
hipMemcpy(h_nnzPerVectorA, d_nnzPerVectorA, n * sizeof(*h_nnzPerVectorA), hipMemcpyDeviceToHost);


int *h_nnzPerVectorEYE = (int *)malloc(n * sizeof(*h_nnzPerVectorEYE));
hipMemcpy(h_nnzPerVectorEYE, d_nnzPerVectorEYE, n * sizeof(*h_nnzPerVectorEYE), hipMemcpyDeviceToHost);

// device side sparse matrix;
double *d_A ; hipMalloc(&d_A, nnzA * sizeof(*d_A));
int *d_A_RowIndices ;hipMalloc(&d_A_RowIndices, (n + 1) * sizeof(*d_A_RowIndices));
int *d_A_ColIndices ;hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices));
hipsparseDdense2csr(handle, n, n, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices);

double *d_EYE ; hipMalloc(&d_EYE, nnzEYE * sizeof(*d_EYE));
int *d_EYE_RowIndices ;hipMalloc(&d_EYE_RowIndices, (n + 1) * sizeof(*d_EYE_RowIndices));
int *d_EYE_ColIndices ;hipMalloc(&d_EYE_ColIndices, nnzEYE * sizeof(*d_EYE_ColIndices));
hipsparseDdense2csr(handle, n, n, descrEYE, d_EYE_dense, lda, d_nnzPerVectorEYE, d_EYE, d_EYE_RowIndices, d_EYE_ColIndices);




// --- Host side sparse matrices
double *h_A = (double *)malloc(nnzA * sizeof(*h_A));

int *h_A_RowIndices = (int *)malloc((n + 1) * sizeof(*h_A_RowIndices));
int *h_A_ColIndices = (int *)malloc(nnzA * sizeof(*h_A_ColIndices));
hipMemcpy(h_A, d_A, nnzA * sizeof(*h_A), hipMemcpyDeviceToHost);
hipMemcpy(h_A_RowIndices, d_A_RowIndices, (n + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost);
hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnzA * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost);



double *h_EYE = (double *)malloc(nnzEYE * sizeof(*h_EYE));

int *h_EYE_RowIndices = (int *)malloc((n + 1) * sizeof(*h_EYE_RowIndices));
int *h_EYE_ColIndices = (int *)malloc(nnzEYE * sizeof(*h_EYE_ColIndices));
hipMemcpy(h_EYE, d_EYE, nnzEYE * sizeof(*h_EYE), hipMemcpyDeviceToHost);
hipMemcpy(h_EYE_RowIndices, d_EYE_RowIndices, (n + 1) * sizeof(*h_EYE_RowIndices), hipMemcpyDeviceToHost);
hipMemcpy(h_EYE_ColIndices, d_EYE_ColIndices, nnzEYE * sizeof(*h_EYE_ColIndices), hipMemcpyDeviceToHost);


irka_iter=0;
//initialize sigma ...
double inii=0.5;double fini=7; 
for(int i=0;i<r;i++){
     sig[i]= log((inii + (fini-inii)*i/r ));
}

double *d_A1_dense;  hipMalloc(&d_A1_dense, n * n * sizeof(*d_A1_dense));
hipsparseMatDescr_t descrA1;    hipsparseCreateMatDescr(&descrA1);
hipsparseSetMatType   (descrA1, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(descrA1, HIPSPARSE_INDEX_BASE_ONE);
int *d_A1_RowIndices;  hipMalloc(&d_A1_RowIndices, (n + 1) * sizeof(*d_A1_RowIndices));
  int *h_A1_RowIndices = (int *)malloc((n + 1) * sizeof(*h_A1_RowIndices));


 int baseA1, nnzA1 = 0;
  // nnzTotalDevHostPtr points to host memory
  int *nnzTotalDevHostPtr = &nnzA1;
 hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST); 


double *h_A1_dense = (double*)malloc(n * n * sizeof(*h_A1_dense));

double *h_A1 = (double *)malloc(nnzA1 * sizeof(*h_A1)); 
//while loop
cout<<"I am Here"<<endl;;

while(error>itol and irka_iter<max_iter){
 irka_iter++; 
cout<<"iteration number"<<irka_iter<<endl;

 for(int i=0;i<n;i++)sig_old[i]=sig[i];


 for (int i=0;i<r;i++){
         
        hipsparseXcsrgeamNnz(handle, n, n, 
                            descrEYE, nnzEYE, d_EYE_RowIndices, d_EYE_ColIndices, 
                            descrA, nnzA, d_A_RowIndices, d_A_ColIndices,
                            descrA1, d_A1_RowIndices, 
                            nnzTotalDevHostPtr);


          if (NULL != nnzTotalDevHostPtr){ nnzA1 = *nnzTotalDevHostPtr; }
          else {
           hipMemcpy(&nnzA1,  d_A1_RowIndices + n, sizeof(int), hipMemcpyDeviceToHost);
           hipMemcpy(&baseA1, d_A1_RowIndices,     sizeof(int), hipMemcpyDeviceToHost);
            nnzA1 -= baseA1;
          }
     

    int *d_A1_ColIndices; hipMalloc(&d_A1_ColIndices, nnzA1 * sizeof(int));
    
 int *h_A1_ColIndices = (int *)malloc(nnzA1 * sizeof(*h_A1_ColIndices));

    
    double *d_A1;         hipMalloc(&d_A1, nnzA1 * sizeof(double));
   
       double alpha; double beta;
       alpha=sig[i];beta=-1;
        
        cout<<"WTF 0"<<endl;
       //////////////////////////////// maybe 
        hipsparseDcsrgeam
                   (handle, n, n,
                    &alpha, 
                    descrEYE, nnzEYE, d_EYE, d_EYE_RowIndices, d_EYE_ColIndices,
                    &beta,
                    descrA, nnzA, d_A, d_A_RowIndices, d_A_ColIndices, 
                    descrA1, d_A1, d_A1_RowIndices, d_A1_ColIndices);
       
       hipsparseDcsr2dense(handle, n, n, descrA1, d_A1, d_A1_RowIndices, d_A1_ColIndices, d_A1_dense, n);
       cout<<"WTF 1"<<endl;
       
       //hipMemcpy(h_A1 ,           d_A1,            nnzA1 * sizeof(double ), hipMemcpyDeviceToHost);
       //hipMemcpy(h_A1_RowIndices, d_A1_RowIndices, (n + 1) * sizeof(*h_A1_RowIndices), hipMemcpyDeviceToHost);
       //hipMemcpy(h_A1_ColIndices, d_A1_ColIndices, nnzA1 * sizeof(*h_A1_ColIndices), hipMemcpyDeviceToHost);
       hipMemcpy(h_A1_dense, d_A1_dense, n * n * sizeof(double), hipMemcpyDeviceToHost);
       //*/
       
       cout<<"WTF 2"<<endl;
        // iintitialise x0 and x0_tilde
        for(int j =0;j<n;j++){
          x0[j]=((rand()+2)%7)/10;x0_tilde[j]=((rand()+7)%10)/10;
        } 
        int status1= bicg(h_A1_dense,x0,B,eye_n,rmaxit,rtol,n);
        int status2= bicg(h_A1_dense,x0_tilde,C,eye_n,rmaxit,rtol,n);

        for(int j=0;j<n;j++){V[j*n+i]=x0[j];W[n*j+i]=x0_tilde[j];}

      cout<<"WTF 3"<<endl;
       hipFree(d_A1);
       hipFree(d_A1_ColIndices);
 
  }
  for(int i=0;i<n;i++){
   for(int j=0;j<r;j++){
    //cout<<V[i*n+j]<<" ";
  }
  //cout<<endl;
}
   
  for(int i=0;i<n;i++){
   for(int j=0;j<r;j++){
    //cout<<W[i*n+j]<<" ";
  }
  //cout<<endl;
}


  // We have V and W matrix .... We need to orthogonalise them ....
  hipsolverHandle_t solver_handle_m;
  hipsolverDnCreate (&solver_handle_m);
 
  hipblasHandle_t cublas_handle_m;
  hipblasCreate(&cublas_handle_m);
  int work_size_m=0;
  int *devInfo_m;
  const int Nrows=n;
  const int Ncols=r;
  double *d_V; hipMalloc(&d_V,Nrows*Ncols*sizeof(double));
  double *d_W; hipMalloc(&d_W,Nrows*Ncols*sizeof(double));
 
 cout<<"Wtf 3-1"<<endl;;
  hipMemcpy(d_V,V,Nrows*Ncols*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_W,W,Nrows*Ncols*sizeof(double), hipMemcpyHostToDevice);
 
  //Cuda Qr initialisation,
  double *d_TAU_V ; hipMalloc(&d_TAU_V,min(Nrows,Ncols)*sizeof(double));
  double *d_TAU_W ; hipMalloc(&d_TAU_W,min(Nrows,Ncols)*sizeof(double));

cout<<"Wtf 3-2"<<endl;;
  hipsolverDnDgeqrf_bufferSize(solver_handle_m, Nrows,Ncols, d_V ,n ,&work_size_m);
cout<<"Wtf 3-3"<<endl;;
  double *work_V_m ; hipMalloc(&work_V_m, work_size_m*sizeof(double ));
  double *work_W_m ; hipMalloc(&work_W_m, work_size_m*sizeof(double ));
  // Cuda GERF exec...
// hipsolverDnDgeqrf_bufferSize(        cusolverH,        m,        n,        d_A,       lda,        &lwork_geqrf);
//cusolver_status = hipsolverDnDgeqrf( cusolverH, m, n, d_A, lda, d_tau, d_work, lwork, devInfo);

cout<<"Wtf 3-4"<<endl;;
  hipsolverDnDgeqrf(solver_handle_m,Nrows,Ncols,d_V,n,d_TAU_V,work_V_m,work_size_m,devInfo_m);
  int devInfo_V_h=0; hipMemcpy(&devInfo_V_h,devInfo_m,sizeof(int),hipMemcpyDeviceToHost);
cout<<"Wtf 3-5"<<endl;;
  hipsolverDnDgeqrf(solver_handle_m, Nrows , Ncols, d_W,n, d_TAU_W, work_W_m , work_size_m,devInfo_m);
  int devInfo_W_h=0; hipMemcpy(&devInfo_W_h,devInfo_m,sizeof(int),hipMemcpyDeviceToHost);
  
  if(devInfo_W_h!=0 or devInfo_V_h!=0){cout<<"Unsuccesful";}
  // At his point the upper triangular part of A contains the elemrnts of R.
cout<<"Wtf 3-6"<<endl;;  

  // Initialising Q matrix.
  double *h_Q_V= (double *)malloc(Nrows*Nrows*sizeof(double));
  double *h_Q_W= (double *)malloc(Nrows*Nrows*sizeof(double));
  for(int j=0;j<Nrows;j++)for(int i=0;i<Nrows;i++){if(j==i){h_Q_V[j+i*Nrows]=1;h_Q_W[j+i*Nrows]=1;}
     
                                              else {h_Q_V[j+i*Nrows]=0;h_Q_W[j+i*Nrows]=0;}  }
  cout<<"wtf 3-7"<<endl;
  double *d_Q_V;hipMalloc(&d_Q_V,Nrows*Nrows*sizeof(double));
  double *d_Q_W;hipMalloc(&d_Q_W,Nrows*Nrows*sizeof(double));
  hipMemcpy(d_Q_V,h_Q_V,Nrows*Nrows*sizeof(double),hipMemcpyHostToDevice);hipMemcpy(d_Q_W,h_Q_W,Nrows*Nrows*sizeof(double),hipMemcpyHostToDevice);
cout<<"Wtf 3-8"<<endl;;
  // CuDA QR execution 
  hipsolverDnDormqr(solver_handle_m,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_N,Nrows,Ncols,min(Nrows,Ncols),d_V,Nrows,d_TAU_V,d_Q_V,Nrows,work_V_m,work_size_m,devInfo_m);
  hipsolverDnDormqr(solver_handle_m,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_N,Nrows,Ncols,min(Nrows,Ncols),d_W,Nrows,d_TAU_W,d_Q_W,Nrows,work_W_m,work_size_m,devInfo_m);
  hipMemcpy(h_Q_V,d_Q_V,Nrows*Nrows*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(h_Q_W,d_Q_W,Nrows*Nrows*sizeof(double), hipMemcpyDeviceToHost);
  for(int i=0;i<n;i++){
    for(int j=0;j<r;j++){
       V[i*n+j]=h_Q_V[i*n+j];
       W[i*n+j]=h_Q_W[i+j*n];// making it W^T

    }
  }
cout<<"Wtf 3-9"<<endl;;
  // V and W have been orthogonalised
  // find Ared , Bred ...
  double *d_Q_V_mod;hipMalloc(&d_Q_V_mod,Nrows*Ncols*sizeof(double));
  double *d_Q_W_mod;hipMalloc(&d_Q_W_mod,Nrows*Ncols*sizeof(double));
  double *d_A_temp; hipMalloc(&d_A_temp,Nrows*Ncols*sizeof(double));
  double *d_A_red; hipMalloc(&d_A_red,Ncols*Ncols*sizeof(double));


  hipMemcpy(d_Q_V_mod,V,Nrows*Ncols*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_Q_W_mod,W,Nrows*Ncols*sizeof(double), hipMemcpyHostToDevice);
cout<<"Wtf 3-10"<<endl;;
  gpu_blas_mmul(d_A_dense, d_Q_V_mod, d_A_temp, n, n, r);// q=Ap
  gpu_blas_mmul(d_Q_W_mod, d_A_temp, d_A_red, r, n, r);

cout<<"Wtf 3-11"<<endl;;
  // d_A_red has the reduced Matrix ....
  double *d_B ;hipMalloc(&d_B ,Nrows*sizeof(double));hipMemcpy(d_B,B,Nrows*sizeof(double),hipMemcpyHostToDevice);
  double *d_B_red ;hipMalloc(&d_B_red ,Ncols*sizeof(double));
  gpu_blas_mmul(d_Q_W, d_B, d_B_red, r, n, 1);
  double *d_C ;hipMalloc(&d_C,Nrows*sizeof(double ));
   double *d_C_red ;hipMalloc(&d_C_red,Ncols*sizeof(double ));
  hipMemcpy(d_C,C,Nrows*sizeof(double),hipMemcpyHostToDevice);
  gpu_blas_mmul(d_C,d_Q_V_mod,d_C_red,1,n,r);
  hipMemcpy(A_red,d_A_red,r*r*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(B_red,d_B_red,r*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(C_red,d_C_red,r*sizeof(double),hipMemcpyDeviceToHost);
 // we find the eiggen values of the Ared ... and change sigma ...
cout<<"Wtf 3-12"<<endl;;   
  double *eigv= (double *)malloc(r*sizeof(double ));
  double *eigvec= (double *)malloc(r*r*sizeof(double ));
cout<<"Wtf 3-12-0"<<endl;;
   
   hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
cout<<"Wtf 3-12-01"<<endl;;   
int  lwork_eig = 0;
   int lda=r;
cout<<"Wtf 3-12-001"<<endl;;
hipsolverHandle_t cusolverH ;hipsolverDnCreate(&cusolverH);
cout<<"Wtf 3-12-1"<<endl;; 
  int  *dev_info_eig;hipMalloc ((void**)&dev_info_eig, sizeof(int));
  double *d_eigv;hipMalloc ((void**)&d_eigv, r*sizeof(double));
  double *d_eigvec;hipMalloc ((void**)&d_eigvec, r*r*sizeof(int));
cout<<"Wtf 3-12-2"<<endl;; 
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
   hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER; 
cout<<"Wtf 3-12-3"<<endl;; 
   cusolver_status = hipsolverDnDsyevd_bufferSize( cusolverH, jobz, uplo, r, d_A_red, lda, d_eigv, &lwork_eig);
cout<<"Wtf 3-12-4"<<endl;; 
   double *d_work_eig;  hipMalloc((void**)&d_work_eig, sizeof(double)*lwork_eig);
   int *devInfo_eig = NULL;
   hipsolverDnDsyevd( cusolverH, jobz, uplo, r, d_A_red, lda, d_eigv, d_work_eig, lwork_eig, devInfo_eig);
cout<<"Wtf 3-12-5"<<endl;; 
   hipMemcpy(eigv,d_eigv,r*sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(eigvec,d_A_red,r*r*sizeof(double),hipMemcpyDeviceToHost);

 cout<<"Wtf 3-13"<<endl;;

  //
   double  norm_sigma=0;error=0;
   for(int j=0;j<r;j++){norm_sigma+=sig[j]*sig[j];sig[j]=eigv[j];error+=(sig[j]-sig_old[j])*(sig[j]-sig_old[j]);}
    error/=norm_sigma;
 cout<<"Wtf 3-14"<<endl;;
 
  hipFree(d_V);
  hipFree(d_W);
  
  hipFree(d_TAU_V);
  hipFree(d_TAU_W);
  
  hipFree(work_V_m);
  hipFree(work_W_m);


  hipFree(d_Q_V);
  hipFree(d_Q_W);
  
  hipFree(d_Q_V_mod);
  hipFree(d_Q_W_mod);

  hipFree(d_A_temp);
  hipFree(d_A_red);
  
  hipFree(d_B);
  hipFree(d_B_red);

  hipFree(d_C);
  hipFree(d_C_red);
  
  hipFree(dev_info_eig);
  hipFree(d_eigv);

  hipFree(d_eigvec);
  hipFree(d_work_eig);

 cout<<"Wtf 3-15"<<endl;;

  free(h_Q_V);
  free(h_Q_W);
  free(B_red);
  free(A_red);
  free(C_red);
  free(eigv);
  free(eigvec);


 cout<<"Wtf 3-16"<<endl;;
  hipsolverDnDestroy(solver_handle_m);
  hipblasDestroy(cublas_handle_m);
  
  cout<<"Wtf 3-17"<<endl;;

 }

for(int i=0;i<r;i++){
   for(int j=0;j<r;j++){
    //cout<<A_red[i*r+j]<<" ";
  }
  cout<<endl;
}

//goes betwwn these two comments.
hipsparseDestroyMatDescr(descrEYE);
hipsparseDestroyMatDescr(descrA);
hipsparseDestroy(handle);
hipFree(d_A1_dense);
hipFree(d_A1_RowIndices);
hipFree(d_EYE);
hipFree(d_EYE_ColIndices);
hipFree(d_EYE_RowIndices);
hipFree(d_A);
hipFree(d_A_RowIndices);
hipFree(d_A_ColIndices);
hipFree(d_A_dense);
hipFree(d_nnzPerVectorA);
hipFree(d_nnzPerVectorEYE);
cout<<"WTF 4"<<endl;


}


